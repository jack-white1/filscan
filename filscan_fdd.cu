#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <sys/time.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

struct header {
    const char *fileName;
    long fileSize;
    long headerSize;
    long dataSize;
    long nsamp;
    long paddedLength;
    uint8_t nbits;
    uint16_t nchans;
    double tsamp;
    double fch1;
    double foff;
};

struct hostFilterbank{
    struct header header;
    uint8_t* data;
};

long get_file_size(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1;
    }

    fseek(file, 0, SEEK_END);
    long size = ftell(file);
    fclose(file);
    return size;
}

long find_header_location(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1;
    }

    const char *search_str = "HEADER_END";
    int search_len = strlen(search_str);
    int match_len = 0;
    long byte_location = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                byte_location = ftell(file) - search_len;
                fclose(file);
                return byte_location;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1;
}

uint8_t find_nbits_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return UINT8_MAX;
    }

    const char *search_str = "nbits";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                uint8_t value;
                if (fread(&value, sizeof(uint8_t), 1, file) == 1) {
                    fclose(file);
                    return value;  // Assumes little-endian order
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return UINT8_MAX;
}

uint16_t find_nchans_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return UINT16_MAX;
    }

    const char *search_str = "nchans";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                uint16_t value;
                if (fread(&value, sizeof(uint16_t), 1, file) == 1) {
                    fclose(file);
                    return value;  // Assumes little-endian order
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return UINT16_MAX;
}

double find_tsamp_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1.0;
    }

    const char *search_str = "tsamp";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                double value;
                if (fread(&value, sizeof(double), 1, file) == 1) {
                    fclose(file);
                    return value;
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1.0;
}

double find_fch1_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1.0;
    }

    const char *search_str = "fch1";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                double value;
                if (fread(&value, sizeof(double), 1, file) == 1) {
                    fclose(file);
                    return value;
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1.0;
}

double find_foff_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1.0;
    }

    const char *search_str = "foff";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                double value;
                if (fread(&value, sizeof(double), 1, file) == 1) {
                    fclose(file);
                    return value;
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1.0;
}

void readHeader(const char* filename, struct header* header){
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return;
    }

    const char *search_str = "HEADER_END";
    int search_len = strlen(search_str);
    int match_len = 0;
    long byte_location = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                byte_location = ftell(file) - search_len;
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fseek(file, 0, SEEK_SET);
    header->fileName = filename;
    header->fileSize = get_file_size(filename);
    header->headerSize = byte_location;
    header->dataSize = header->fileSize - header->headerSize;
    header->nbits = find_nbits_value(filename);
    header->nchans = find_nchans_value(filename);
    header->tsamp = find_tsamp_value(filename);
    header->nsamp = header->dataSize / header->nchans;
    header->fch1 = find_fch1_value(filename);
    header->foff = find_foff_value(filename);

    long nextPowerOf2 = 1;
    while (nextPowerOf2 < header->nsamp) {
        nextPowerOf2 *= 2;
    }

    header->paddedLength = nextPowerOf2;

    fclose(file);
}

void printHeaderStruct(struct header* header){
    printf("File name:\t\t\t%s\n", header->fileName);
    printf("Total file size:\t\t%ld bytes\n", header->fileSize);
    printf("Header size:\t\t\t%ld bytes\n", header->headerSize);
    printf("Data size:\t\t\t%ld bytes\n", header->dataSize);
    printf("nbits:\t\t\t\t%d\n", header->nbits);
    printf("nchans:\t\t\t\t%d\n", header->nchans);
    printf("fch1:\t\t\t\t%lf\n", header->fch1);
    printf("foff:\t\t\t\t%lf\n", header->foff);
    printf("tsamp:\t\t\t\t%lf\n", header->tsamp);
    printf("nsamp:\t\t\t\t%ld\n", header->nsamp);
    printf("True observation time:\t\t%lf s\n", header->tsamp * header->nsamp);
}

void readFilterbankData(struct header* header, struct hostFilterbank* hostFilterbank){
    FILE *file = fopen(header->fileName, "rb");
    if (!file) {
        perror("Error opening file");
        return;
    }

    fseek(file, header->headerSize, SEEK_SET);
    fread(hostFilterbank->data, sizeof(uint8_t), header->dataSize, file);
    fclose(file);
}

__global__ void transpose_and_cast_uint8_t_to_padded_float(uint8_t* deviceData_uint8_t, float* deviceData_float, int nchans, int input_nsamps, int output_nsamps) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < input_nsamps && y < nchans) {
        deviceData_float[y * output_nsamps + x] = (float) deviceData_uint8_t[x * nchans + y];
    }
}

static __constant__ float cachedTimeShiftsPerDM[4096];

__global__ void rotate_spectrum(float2* inputArray, float2* outputArray, long nchans, long nsamps, float DM){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y;

    int outputIndex = y * nsamps + x;

    if (x < nsamps && y < nchans) {
        float phase = x * DM * cachedTimeShiftsPerDM[y];
        float2 input = inputArray[outputIndex];
        float2 output;
        float s, c;
        sincosf(phase, &s, &c);
        output.x = input.x * c - input.y * s;
        output.y = input.x * s + input.y * c;
        outputArray[outputIndex] = output;
    }
}


__global__ void sum_across_channels(float2* inputArray, float2* outputArray, long nchans, long nsamps){
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    float2 sum;
    sum.x = 0.0;
    sum.y = 0.0;

    if (x < nsamps) {
        for (int y = 0; y < nchans; y++) {
            float2 value = inputArray[y * nsamps + x];
            sum.x += value.x;
            sum.y += value.y;
        }
    }
    
    outputArray[x] = sum;
}

void compute_time_shifts(float* timeShifts, float f1, float foff, int nchans, float DM, float FFTbinWidth) {
    for (int i = 0; i < nchans; i++) {
        float f2 = f1 + foff * i;

        // convert to GHz
        float f1_GHz = f1 / 1000.0;
        float f2_GHz = f2 / 1000.0;
        float k = 4.148808;

        // compute the time shift in ms
        float timeShift_ms = k * DM * (1.0 / (f1_GHz * f1_GHz) - 1.0 / (f2_GHz * f2_GHz));

        // convert to seconds
        timeShifts[i] = - timeShift_ms / 1000.0;
        timeShifts[i] *= 2.0 * M_PI * FFTbinWidth;
    }
}


const char* filscan_frame = 

"   ______________ __                    \n"
"    _____  ____(_) /_____________ _____ \n"
"     ___  /_  / / / ___/ ___/ __ `/ __ \\ \n"
"      _  __/ / / (__  ) /__/ /_/ / / / /\n"
"      /_/   /_/_/____/\\___/\\__,_/_/ /_/\n\n";



int main(int argc, char *argv[]) {
    printf("%s", filscan_frame);

    if (argc != 2) {
        printf("Usage: %s <file_name>\n", argv[0]);
        return 1;
    }

    // initialise error, timing and available memory variables for use throughout the program
    hipError_t error = hipGetLastError();
    struct timeval start, end;
    gettimeofday(&start, NULL);
    size_t availableMemory, totalMemory;

    // print the available memory on the GPU
    hipMemGetInfo(&availableMemory, &totalMemory);
    printf("Available memory:\t\t%ld MB\n", availableMemory / 1024 / 1024);
    printf("Total memory:\t\t\t%ld MB\n", totalMemory / 1024 / 1024);

    struct header header;
    readHeader(argv[1], &header);
    printHeaderStruct(&header);

    struct hostFilterbank hostFilterbank;
    hostFilterbank.header = header;
    hostFilterbank.data = (uint8_t*) malloc(header.dataSize * sizeof(uint8_t));
    readFilterbankData(&header, &hostFilterbank);

    printf("Next power of 2:\t\t%ld\n", header.paddedLength);
    printf("Padded observation time:\t%lf\n", header.tsamp * header.paddedLength);
    printf("FFT bin width\t\t\t%lf Hz\n", 1.0 / (header.tsamp * header.paddedLength));

    float FFTbinWidth = 1.0 / (header.tsamp * header.paddedLength);

    printf("Data length:\t\t\t%ld bytes\n", header.nchans * header.paddedLength);

    // allocate memory on the device
    u_int8_t* deviceData_uint8_t;
    float* deviceData_float;
    float2* deviceData_float2_raw;
    float2* deviceData_float2_dedispersed;
    float2* deviceData_float2_single_spectrum;

    hipMalloc((void**)&deviceData_uint8_t, header.dataSize * sizeof(uint8_t));
    hipMalloc((void**)&deviceData_float, header.nchans * header.paddedLength * sizeof(float));
    hipMalloc((void**)&deviceData_float2_raw, ((header.paddedLength/2)+1) * header.nchans * sizeof(float2));
    hipMalloc((void**)&deviceData_float2_dedispersed, ((header.paddedLength/2)+1) * header.nchans * sizeof(float2));
    hipMalloc((void**)&deviceData_float2_single_spectrum, ((header.paddedLength/2)+1) * sizeof(float2));
    hipMemGetInfo(&availableMemory, &totalMemory);
    printf("\nAvailable memory after first mallocs:\t\t%ld MB\n", availableMemory / 1024 / 1024);

    hipMemset(deviceData_float, 0, header.nchans * header.paddedLength * sizeof(float));

    hipMemcpy(deviceData_uint8_t, hostFilterbank.data, header.dataSize * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipDeviceSynchronize();


    // transpose and cast
    dim3 dimBlock(32, 32);
    dim3 dimGrid((header.paddedLength + dimBlock.x - 1) / dimBlock.x, (header.nchans + dimBlock.y - 1) / dimBlock.y);
    transpose_and_cast_uint8_t_to_padded_float<<<dimGrid, dimBlock>>>(deviceData_uint8_t, deviceData_float, header.nchans, header.nsamp, header.paddedLength);
    hipDeviceSynchronize();
    hipFree(deviceData_uint8_t);
    hipDeviceSynchronize();
    hipMemGetInfo(&availableMemory, &totalMemory);
    printf("Available memory after free uint8:\t\t%ld MB\n", availableMemory / 1024 / 1024);


    // cufft each channel, storing the output in the float2 array
    hipfftHandle plan;
    hipfftPlan1d(&plan, header.paddedLength, HIPFFT_R2C, header.nchans);
    hipfftExecR2C(plan, deviceData_float, deviceData_float2_raw);
    hipDeviceSynchronize();
    hipFree(deviceData_float);
    hipDeviceSynchronize();
    hipMemGetInfo(&availableMemory, &totalMemory);
    printf("Available memory after free float:\t\t%ld MB\n", availableMemory / 1024 / 1024);

    // compute the time shifts for each channel
    float* timeShifts = (float*) malloc(header.nchans * sizeof(float));
    compute_time_shifts(timeShifts, header.fch1, header.foff, header.nchans, 1.0, FFTbinWidth);
    hipMemcpyToSymbol(HIP_SYMBOL(cachedTimeShiftsPerDM), timeShifts, header.nchans * sizeof(float));

    float* deviceTimeShifts;
    hipMalloc((void**)&deviceTimeShifts, header.nchans * sizeof(float));
    hipMemcpy(deviceTimeShifts, timeShifts, header.nchans * sizeof(float), hipMemcpyHostToDevice);

    float DM = 0;
    float DM_step = 0.1;

    for (int DM_idx = 0; DM_idx < 1024; DM_idx++){
        DM += DM_step;

        // time the kernel
        hipEvent_t startKernel, stopKernel;
        hipEventCreate(&startKernel);
        hipEventCreate(&stopKernel);
        hipEventRecord(startKernel, 0);

        // rotate the spectrum
        dim3 dimBlockRotation(1024, 1);
        dim3 dimGridRotation((header.paddedLength + dimBlockRotation.x - 1) / dimBlockRotation.x, header.nchans);
        rotate_spectrum<<<dimGridRotation, dimBlockRotation>>>(deviceData_float2_raw, deviceData_float2_dedispersed, header.nchans, header.paddedLength, DM);
        hipDeviceSynchronize();

        // stop timing
        hipEventRecord(stopKernel, 0);
        hipEventSynchronize(stopKernel);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, startKernel, stopKernel);
        printf("\nRotation kernel time:\t\t\t%lf s\n", elapsedTime / 1000.0);


        // time the kernel
        hipEvent_t startKernel2, stopKernel2;
        hipEventCreate(&startKernel2);
        hipEventCreate(&stopKernel2);
        hipEventRecord(startKernel2, 0);

        // sum across channels
        dim3 dimBlockSum(1024, 1);
        dim3 dimGridSum((header.paddedLength + dimBlockSum.x - 1) / dimBlockSum.x);
        sum_across_channels<<<dimGridSum, dimBlockSum>>>(deviceData_float2_dedispersed, deviceData_float2_single_spectrum, header.nchans, header.paddedLength);
        hipDeviceSynchronize();

        // stop timing
        hipEventRecord(stopKernel2, 0);
        hipEventSynchronize(stopKernel2);
        float elapsedTime2;
        hipEventElapsedTime(&elapsedTime2, startKernel2, stopKernel2);
        printf("Sum kernel time:\t\t\t%lf s\n", elapsedTime2 / 1000.0);
    }

    // free memory
    hipFree(deviceData_float2_raw);
    hipFree(deviceData_float2_dedispersed);
    hipFree(deviceData_float2_single_spectrum);
    free(hostFilterbank.data);
    free(timeShifts);

    // check cuda error
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error 3: %s\n", hipGetErrorString(error));
        return 1;
    }

    // stop timing
    gettimeofday(&end, NULL);
    printf("Total time:\t\t\t%lf s\n", (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0);
    return 0;
}
