#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <sys/time.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define ROTATION_SMEM_WIDTH 128
#define ROTATION_SMEM_HEIGHT 16

struct header {
    const char *fileName;
    long fileSize;
    long headerSize;
    long dataSize;
    long nsamp;
    uint8_t nbits;
    uint16_t nchans;
    double tsamp;
    double fch1;
    double foff;
};

struct hostFilterbank{
    struct header header;
    uint8_t* data;
};

long get_file_size(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1;
    }

    fseek(file, 0, SEEK_END);
    long size = ftell(file);
    fclose(file);
    return size;
}

long find_header_location(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1;
    }

    const char *search_str = "HEADER_END";
    int search_len = strlen(search_str);
    int match_len = 0;
    long byte_location = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                byte_location = ftell(file) - search_len;
                fclose(file);
                return byte_location;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1;
}

uint8_t find_nbits_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return UINT8_MAX;
    }

    const char *search_str = "nbits";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                uint8_t value;
                if (fread(&value, sizeof(uint8_t), 1, file) == 1) {
                    fclose(file);
                    return value;  // Assumes little-endian order
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return UINT8_MAX;
}

uint16_t find_nchans_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return UINT16_MAX;
    }

    const char *search_str = "nchans";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                uint16_t value;
                if (fread(&value, sizeof(uint16_t), 1, file) == 1) {
                    fclose(file);
                    return value;  // Assumes little-endian order
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return UINT16_MAX;
}

double find_tsamp_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1.0;
    }

    const char *search_str = "tsamp";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                double value;
                if (fread(&value, sizeof(double), 1, file) == 1) {
                    fclose(file);
                    return value;
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1.0;
}

double find_fch1_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1.0;
    }

    const char *search_str = "fch1";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                double value;
                if (fread(&value, sizeof(double), 1, file) == 1) {
                    fclose(file);
                    return value;
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1.0;
}

double find_foff_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1.0;
    }

    const char *search_str = "foff";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                double value;
                if (fread(&value, sizeof(double), 1, file) == 1) {
                    fclose(file);
                    return value;
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1.0;
}

void readHeader(const char* filename, struct header* header){
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return;
    }

    const char *search_str = "HEADER_END";
    int search_len = strlen(search_str);
    int match_len = 0;
    long byte_location = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                byte_location = ftell(file) - search_len;
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fseek(file, 0, SEEK_SET);
    header->fileName = filename;
    header->fileSize = get_file_size(filename);
    header->headerSize = byte_location;
    header->dataSize = header->fileSize - header->headerSize;
    header->nbits = find_nbits_value(filename);
    header->nchans = find_nchans_value(filename);
    header->tsamp = find_tsamp_value(filename);
    header->nsamp = header->dataSize / header->nchans;
    header->fch1 = find_fch1_value(filename);
    header->foff = find_foff_value(filename);

    fclose(file);
}

void printHeaderStruct(struct header* header){
    printf("File name:\t\t\t%s\n", header->fileName);
    printf("Total file size:\t\t%ld bytes\n", header->fileSize);
    printf("Header size:\t\t\t%ld bytes\n", header->headerSize);
    printf("Data size:\t\t\t%ld bytes\n", header->dataSize);
    printf("nbits:\t\t\t\t%d\n", header->nbits);
    printf("nchans:\t\t\t\t%d\n", header->nchans);
    printf("fch1:\t\t\t\t%lf\n", header->fch1);
    printf("foff:\t\t\t\t%lf\n", header->foff);
    printf("tsamp:\t\t\t\t%lf\n", header->tsamp);
    printf("nsamp:\t\t\t\t%ld\n", header->nsamp);
    printf("True observation time:\t\t%lf s\n", header->tsamp * header->nsamp);
}

void readFilterbankData(struct header* header, struct hostFilterbank* hostFilterbank){
    FILE *file = fopen(header->fileName, "rb");
    if (!file) {
        perror("Error opening file");
        return;
    }

    fseek(file, header->headerSize, SEEK_SET);
    fread(hostFilterbank->data, sizeof(uint8_t), header->dataSize, file);
    fclose(file);
}

__global__ void transpose_and_cast_uint8_t_to_padded_float(uint8_t* deviceData_uint8_t, float* deviceData_float, int nchans, int input_nsamps, int output_nsamps) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < input_nsamps && y < nchans) {
        deviceData_float[y * output_nsamps + x] = (float) deviceData_uint8_t[x * nchans + y];
    }
}

static __constant__ float cachedTimeShiftsPerDM[4096];

struct SharedMemory2D {
    float2* data;
    __device__ float2* operator[](int idx) {
        return &data[idx * ROTATION_SMEM_WIDTH];
    }
};

__global__ void rotate_spectrum_smem(float2* deviceData_float2, float2* deviceData_output_float2, long nsamps, float FFTbinWidth, long nchans, float DMstart, float DMstep){
    //extern __shared__ float2 input[ROTATION_SMEM_HEIGHT][ROTATION_SMEM_WIDTH];   // ROTATION_SMEM_HEIGHT channels, ROTATION_SMEM_WIDTH samples per channel
    //extern __shared__ float2 output[ROTATION_SMEM_HEIGHT][ROTATION_SMEM_WIDTH];  // ROTATION_SMEM_HEIGHT DMs, ROTATION_SMEM_WIDTH samples per channel

    extern __shared__ float2 sharedMemory[];

    SharedMemory2D input = { &sharedMemory[0] };
    SharedMemory2D output = { &sharedMemory[ROTATION_SMEM_HEIGHT * ROTATION_SMEM_WIDTH] };  // Offset by the size of the input array

    // threadIdx.x = 0 -> ROTATION_SMEM_WIDTH-1
    // threadIdx.y = 0
    
    // blockDim.x = ROTATION_SMEM_WIDTH
    // blockDim.y = 1

    // gridDim.x = nsamps / ROTATION_SMEM_WIDTH = ((nextPowerOf2/2)+1) / ROTATION_SMEM_WIDTH
    // gridDim.y = nchans / ROTATION_SMEM_HEIGHT

    // starting channel = blockIdx.y * ROTATION_SMEM_HEIGHT

    // copy channel 0 -> ROTATION_SMEM_HEIGHT - 1 data to shared memory, ROTATION_SMEM_WIDTH samples wide
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * ROTATION_SMEM_HEIGHT;

    if (global_x < nsamps && global_y < nchans){
        for (int y_i = 0; y_i < ROTATION_SMEM_HEIGHT; y_i++){
            input[y_i][threadIdx.x] = deviceData_float2[(global_y + y_i) * nsamps + global_x];       
        }
    }

    // set output to 0
    for (int y_i = 0; y_i < ROTATION_SMEM_HEIGHT; y_i++){
        output[y_i][threadIdx.x].x = 0.0f;
        output[y_i][threadIdx.x].y = 0.0f;
    }

    __syncthreads();

    float multiplier = 2.0 * M_PI * FFTbinWidth * global_x;
    float DM;

    // initialise rotations
    // rotate channel 0 -> ROTATION_SMEM_HEIGHT - 1 data in shared memory, ROTATION_SMEM_WIDTH samples wide
    for (int DM_i = 0; DM_i < ROTATION_SMEM_HEIGHT; DM_i++){
        DM = DMstart + DM_i * DMstep;
        if (global_x < nsamps && global_y < nchans){
            float2 value;
            float phase;
            float s, c;
            for (int y_i = 0; y_i < ROTATION_SMEM_HEIGHT; y_i++){
                value = input[y_i][threadIdx.x];
                phase = multiplier * cachedTimeShiftsPerDM[global_y + y_i] * DM;
                __sincosf(phase, &s, &c);
                input[y_i][threadIdx.x].x += value.x * c - value.y * s;
                input[y_i][threadIdx.x].y += value.x * s + value.y * c;
                output[DM_i][threadIdx.x].x += input[y_i][threadIdx.x].x;
                output[DM_i][threadIdx.x].y += input[y_i][threadIdx.x].y;
            }
        }
    }

    __syncthreads();



    // copy channel 0 -> ROTATION_SMEM_HEIGHT - 1 data from shared memory to global memory, ROTATION_SMEM_WIDTH samples wide
    if (global_x < nsamps && global_y < nchans){
        for (int y_i = 0; y_i < ROTATION_SMEM_HEIGHT; y_i++){
            deviceData_output_float2[(global_y + y_i) * nsamps + global_x] = output[y_i][threadIdx.x];  // overreaching?
        }
    }
}

void compute_time_shifts(float* timeShifts, float f1, float foff, int nchans, float DM) {
    for (int i = 0; i < nchans; i++) {
        float f2 = f1 + foff * i;

        // convert to GHz
        float f1_GHz = f1 / 1000.0;
        float f2_GHz = f2 / 1000.0;
        float k = 4.148808;

        // compute the time shift in ms
        float timeShift_ms = k * DM * (1.0 / (f1_GHz * f1_GHz) - 1.0 / (f2_GHz * f2_GHz));

        // convert to seconds
        timeShifts[i] = - timeShift_ms / 1000.0;
    }
}


const char* filscan_frame = 

"   ______________ __                    \n"
"    _____  ____(_) /_____________ _____ \n"
"     ___  /_  / / / ___/ ___/ __ `/ __ \\ \n"
"      _  __/ / / (__  ) /__/ /_/ / / / /\n"
"      /_/   /_/_/____/\\___/\\__,_/_/ /_/\n\n";



int main(int argc, char *argv[]) {
    // begin overall program timer using gettimeofday()
    struct timeval start, end;
    gettimeofday(&start, NULL);

    // begin load data timer
    struct timeval load_start, load_end;
    gettimeofday(&load_start, NULL);

    printf("%s", filscan_frame);

    if (argc != 2) {
        printf("Usage: %s <file_name>\n", argv[0]);
        return 1;
    }


    struct header header;
    readHeader(argv[1], &header);
    printHeaderStruct(&header);

    struct hostFilterbank hostFilterbank;
    hostFilterbank.header = header;
    hostFilterbank.data = (uint8_t*) malloc(header.dataSize * sizeof(uint8_t));
    readFilterbankData(&header, &hostFilterbank);

    // end load data timer using gettimeofday()
    gettimeofday(&load_end, NULL);
    double load_elapsed = (load_end.tv_sec - load_start.tv_sec) + (load_end.tv_usec - load_start.tv_usec) / 1000000.0;



    // initialise all GPU arrays here
    // start cuda malloc timer using gettimeofday()
    struct timeval malloc_start, malloc_end;
    gettimeofday(&malloc_start, NULL);

    long nsamps = (long) header.nsamp;
    long nextPowerOf2 = 1;
    while (nextPowerOf2 < nsamps) {
        nextPowerOf2 *= 2;
    }
    printf("Next power of 2:\t\t%ld\n", nextPowerOf2);
    printf("Padded observation time:\t%lf\n", header.tsamp * nextPowerOf2);
    printf("FFT bin width\t\t\t%lf Hz\n", 1.0 / (header.tsamp * nextPowerOf2));

    float FFTbinWidth = 1.0 / (header.tsamp * nextPowerOf2);

    long nchans = (long) header.nchans;
    long dataLength = nchans * nextPowerOf2;

    printf("Data length:\t\t\t%ld bytes\n", dataLength);

    u_int8_t* deviceData_uint8_t;
    hipMalloc((void**)&deviceData_uint8_t, header.dataSize * sizeof(uint8_t));

    float* deviceData_float;
    hipMalloc((void**)&deviceData_float, dataLength * sizeof(float));

    // check errors after mallocs
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("device data CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipMemset(deviceData_float, 0, dataLength * sizeof(float));

    // make a float2 copy of the array for cufft output
    float2* deviceData_float2;
    hipMalloc((void**)&deviceData_float2, ((nextPowerOf2/2)+1) * nchans * sizeof(float2));

    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("float2 CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }





    // print memory utilisation statistics
    size_t free_byte;
    size_t total_byte;
    hipMemGetInfo(&free_byte, &total_byte);
    hipMemGetInfo(&free_byte, &total_byte);
    double free_db = (double)free_byte;
    double total_db = (double)total_byte;
    double used_db = total_db - free_db;
    printf("\nGPU memory usage:\t\tused = %f, free = %f MB, total = %f MB\n", used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);


    float2* deviceData_float2_summed;
    hipMalloc((void**)&deviceData_float2_summed, ((nextPowerOf2/2)+1) * sizeof(float2));

    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("malloc2 CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }


    // end cuda malloc timer using gettimeofday()
    gettimeofday(&malloc_end, NULL);

    // start data transfer timer using gettimeofday()
    struct timeval transfer_start, transfer_end;
    gettimeofday(&transfer_start, NULL);

    hipMemcpy(deviceData_uint8_t, hostFilterbank.data, header.dataSize * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("H2D CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // end data transfer timer using gettimeofday()
    gettimeofday(&transfer_end, NULL);

    // start transpose cast timer using gettimeofday()
    struct timeval transpose_start, transpose_end;
    gettimeofday(&transpose_start, NULL);

    // transpose and cast
    // copy into float array where each channel is padded to the next highest power of 2 length and set to 0


    dim3 dimBlock(32, 32);
    dim3 dimGrid((nsamps + dimBlock.x - 1) / dimBlock.x, (nchans + dimBlock.y - 1) / dimBlock.y);
    transpose_and_cast_uint8_t_to_padded_float<<<dimGrid, dimBlock>>>(deviceData_uint8_t, deviceData_float, nchans, nsamps, nextPowerOf2);
    hipDeviceSynchronize();
    hipFree(deviceData_uint8_t);
    hipDeviceSynchronize();

    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("transpose CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
        
    // end transpose cast timer using gettimeofday()
    gettimeofday(&transpose_end, NULL);


    // start FFT timer using gettimeofday()
    struct timeval fft_start, fft_end;
    gettimeofday(&fft_start, NULL);

    // cufft each channel, storing the output in the float2 array
    hipfftHandle plan;
    hipfftPlan1d(&plan, nextPowerOf2, HIPFFT_R2C, nchans);
    hipfftExecR2C(plan, deviceData_float, deviceData_float2);
    hipDeviceSynchronize();
    hipFree(deviceData_float);
    hipDeviceSynchronize();

    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("cufft CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }


    // end FFT timer using gettimeofday()
    gettimeofday(&fft_end, NULL);
    
    // start rotation timer using gettimeofday()
    struct timeval rotate_start, rotate_end;
    gettimeofday(&rotate_start, NULL);

    // compute the time shifts for each channel
    float* timeShifts = (float*) malloc(nchans * sizeof(float));
    compute_time_shifts(timeShifts, header.fch1, header.foff, nchans, 1.0);

    // copy the time shifts to the device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(cachedTimeShiftsPerDM), timeShifts, nchans * sizeof(float));
    
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("symbol CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    float DM = 50;

    // rotate the spectrum channelwise
    
    float DMstep = 0.1;

    dim3 dimBlock2(ROTATION_SMEM_WIDTH, 1);
    dim3 dimGrid2(((nextPowerOf2/2)+1 + dimBlock2.x - 1) / dimBlock2.x, nchans / ROTATION_SMEM_HEIGHT);

    hipDeviceSynchronize();
    // get last cuda error
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("before kernel CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    printf("Launching kernel with arguments:\n");
    printf("dimGrid2: %d, %d\n", dimGrid2.x, dimGrid2.y);
    printf("dimBlock2: %d, %d\n", dimBlock2.x, dimBlock2.y);
    printf("smem: %ld\n", 2 * ROTATION_SMEM_HEIGHT * ROTATION_SMEM_WIDTH * sizeof(float2));
    printf("deviceData_float2: %p\n", deviceData_float2);
    printf("deviceData_float2: %p\n", deviceData_float2);
    printf("((nextPowerOf2/2)+1): %ld\n", ((nextPowerOf2/2)+1));
    printf("FFTbinWidth: %f\n", FFTbinWidth);
    printf("nchans: %ld\n", nchans);
    printf("DM: %f\n", DM);
    printf("DMstep: %f\n", DMstep);

    hipFuncSetAttribute(reinterpret_cast<const void*>(rotate_spectrum_smem), hipFuncAttributeMaxDynamicSharedMemorySize, 99*1024);
    rotate_spectrum_smem<<<dimGrid2, dimBlock2, 2 * ROTATION_SMEM_HEIGHT * ROTATION_SMEM_WIDTH * sizeof(float2)>>>(deviceData_float2, deviceData_float2, ((nextPowerOf2/2)+1), FFTbinWidth, nchans, DM, DMstep);
    hipDeviceSynchronize();
    
    // get last cuda error
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("after kernel CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // end rotation timer using gettimeofday()
    gettimeofday(&rotate_end, NULL);

    // start free memory timer using gettimeofday()
    struct timeval free_start, free_end;
    gettimeofday(&free_start, NULL);

    // free memory
    hipFree(deviceData_uint8_t);
    hipFree(deviceData_float);
    hipFree(deviceData_float2);
    //cudaFree(deviceTimeShifts);
    free(hostFilterbank.data);
    free(timeShifts);

    // end free memory timer using gettimeofday()
    gettimeofday(&free_end, NULL);

    // end overall program timer using gettimeofday()
    gettimeofday(&end, NULL);
    double elapsed = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;

    // print timing statistics
    printf("\nLoad data time:\t\t\t%lf s\n", load_elapsed);
    printf("Malloc time:\t\t\t%lf s\n", (malloc_end.tv_sec - malloc_start.tv_sec) + (malloc_end.tv_usec - malloc_start.tv_usec) / 1000000.0);
    printf("Data transfer time H2D:\t\t%lf s\n", (transfer_end.tv_sec - transfer_start.tv_sec) + (transfer_end.tv_usec - transfer_start.tv_usec) / 1000000.0);
    printf("Transpose and cast time:\t%lf s\n", (transpose_end.tv_sec - transpose_start.tv_sec) + (transpose_end.tv_usec - transpose_start.tv_usec) / 1000000.0);
    printf("FFT time:\t\t\t%lf s\n", (fft_end.tv_sec - fft_start.tv_sec) + (fft_end.tv_usec - fft_start.tv_usec) / 1000000.0);
    printf("Rotation time:\t\t\t%lf s\n", (rotate_end.tv_sec - rotate_start.tv_sec) + (rotate_end.tv_usec - rotate_start.tv_usec) / 1000000.0);
    printf("Free memory time:\t\t%lf s\n", (free_end.tv_sec - free_start.tv_sec) + (free_end.tv_usec - free_start.tv_usec) / 1000000.0);
    printf("\nTotal elapsed time:\t\t%lf s\n", elapsed);


    return 0;
}
