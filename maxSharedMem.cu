#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void my_kernel() {
    extern __shared__ float sharedMem[];
    // Do something with shared memory if needed
}

int main() {

    // Set the maximum dynamic shared memory size for the kernel
    size_t desiredSharedMemSize = 99 * 1024;  // 96KB as an example
    hipError_t err = hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to set the attribute: %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipFuncSetAttribute(reinterpret_cast<const void*>(my_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, desiredSharedMemSize);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to set the attribute: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Now, let's retrieve the set value to verify
    hipFuncAttributes attributes;
    err = hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(my_kernel));

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get function attributes: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Maximum dynamic shared memory size for 'my_kernel': %d bytes\n", attributes.maxDynamicSharedSizeBytes);

    return 0;
}
