#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <sys/time.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

struct header {
    const char *fileName;
    long fileSize;
    long headerSize;
    long dataSize;
    long nsamp;
    long paddedLength;
    uint8_t nbits;
    uint16_t nchans;
    double tsamp;
    double fch1;
    double foff;
};

struct hostFilterbank{
    struct header header;
    uint8_t* data;
};

long get_file_size(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1;
    }

    fseek(file, 0, SEEK_END);
    long size = ftell(file);
    fclose(file);
    return size;
}

long find_header_location(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1;
    }

    const char *search_str = "HEADER_END";
    int search_len = strlen(search_str);
    int match_len = 0;
    long byte_location = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                byte_location = ftell(file) - search_len;
                fclose(file);
                return byte_location;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1;
}

uint8_t find_nbits_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return UINT8_MAX;
    }

    const char *search_str = "nbits";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                uint8_t value;
                if (fread(&value, sizeof(uint8_t), 1, file) == 1) {
                    fclose(file);
                    return value;  // Assumes little-endian order
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return UINT8_MAX;
}

uint16_t find_nchans_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return UINT16_MAX;
    }

    const char *search_str = "nchans";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                uint16_t value;
                if (fread(&value, sizeof(uint16_t), 1, file) == 1) {
                    fclose(file);
                    return value;  // Assumes little-endian order
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return UINT16_MAX;
}

double find_tsamp_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1.0;
    }

    const char *search_str = "tsamp";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                double value;
                if (fread(&value, sizeof(double), 1, file) == 1) {
                    fclose(file);
                    return value;
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1.0;
}

double find_fch1_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1.0;
    }

    const char *search_str = "fch1";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                double value;
                if (fread(&value, sizeof(double), 1, file) == 1) {
                    fclose(file);
                    return value;
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1.0;
}

double find_foff_value(const char *filename) {
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return -1.0;
    }

    const char *search_str = "foff";
    int search_len = strlen(search_str);
    int match_len = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                double value;
                if (fread(&value, sizeof(double), 1, file) == 1) {
                    fclose(file);
                    return value;
                }
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fclose(file);
    return -1.0;
}

void readHeader(const char* filename, struct header* header){
    FILE *file = fopen(filename, "rb");
    if (!file) {
        perror("Error opening file");
        return;
    }

    const char *search_str = "HEADER_END";
    int search_len = strlen(search_str);
    int match_len = 0;
    long byte_location = 0;

    char ch;
    while (fread(&ch, 1, 1, file) == 1) {
        if (ch == search_str[match_len]) {
            match_len++;
            if (match_len == search_len) {
                byte_location = ftell(file) - search_len;
                break;
            }
        } else {
            match_len = 0;
        }
    }

    fseek(file, 0, SEEK_SET);
    header->fileName = filename;
    header->fileSize = get_file_size(filename);
    header->headerSize = byte_location;
    header->dataSize = header->fileSize - header->headerSize;
    header->nbits = find_nbits_value(filename);
    header->nchans = find_nchans_value(filename);
    header->tsamp = find_tsamp_value(filename);
    header->nsamp = header->dataSize / header->nchans;
    header->fch1 = find_fch1_value(filename);
    header->foff = find_foff_value(filename);

    long nextPowerOf2 = 1;
    while (nextPowerOf2 < header->nsamp) {
        nextPowerOf2 *= 2;
    }

    header->paddedLength = nextPowerOf2;

    fclose(file);
}

void printHeaderStruct(struct header* header){
    printf("File name:\t\t\t%s\n", header->fileName);
    printf("Total file size:\t\t%ld bytes\n", header->fileSize);
    printf("Header size:\t\t\t%ld bytes\n", header->headerSize);
    printf("Data size:\t\t\t%ld bytes\n", header->dataSize);
    printf("nbits:\t\t\t\t%d\n", header->nbits);
    printf("nchans:\t\t\t\t%d\n", header->nchans);
    printf("fch1:\t\t\t\t%lf\n", header->fch1);
    printf("foff:\t\t\t\t%lf\n", header->foff);
    printf("tsamp:\t\t\t\t%lf\n", header->tsamp);
    printf("nsamp:\t\t\t\t%ld\n", header->nsamp);
    printf("True observation time:\t\t%lf s\n", header->tsamp * header->nsamp);
}

void readFilterbankData(struct header* header, struct hostFilterbank* hostFilterbank){
    FILE *file = fopen(header->fileName, "rb");
    if (!file) {
        perror("Error opening file");
        return;
    }

    fseek(file, header->headerSize, SEEK_SET);
    fread(hostFilterbank->data, sizeof(uint8_t), header->dataSize, file);
    fclose(file);
}

void compute_time_shifts(double* timeShifts, double f1, double foff, int nchans, double DM, double FFTbinWidth) {
    for (int i = 0; i < nchans; i++) {
        double f2 = f1 + foff * i;

        // convert to GHz
        double f1_GHz = f1 / 1000.0;
        double f2_GHz = f2 / 1000.0;
        double k = 4.148808;

        // compute the time shift in ms
        double timeShift_ms = k * DM * (1.0 / (f1_GHz * f1_GHz) - 1.0 / (f2_GHz * f2_GHz));

        // convert to seconds
        timeShifts[i] = - timeShift_ms / 1000.0;
    }
}



const char* filscan_frame = 

"   ______________ __                    \n"
"    _____  ____(_) /_____________ _____ \n"
"     ___  /_  / / / ___/ ___/ __ `/ __ \\ \n"
"      _  __/ / / (__  ) /__/ /_/ / / / /\n"
"      /_/   /_/_/____/\\___/\\__,_/_/ /_/\n\n";



int main(int argc, char *argv[]) {
    printf("%s", filscan_frame);

    if (argc != 2) {
        printf("Usage: %s <file_name>\n", argv[0]);
        return 1;
    }

    // initialise error, timing and available memory variables for use throughout the program
    hipError_t error = hipGetLastError();
    struct timeval start, end;
    gettimeofday(&start, NULL);

    struct header header;
    readHeader(argv[1], &header);
    printHeaderStruct(&header);

    struct hostFilterbank hostFilterbank;
    hostFilterbank.header = header;
    hostFilterbank.data = (uint8_t*) malloc(header.dataSize * sizeof(uint8_t));
    readFilterbankData(&header, &hostFilterbank);

    printf("Next power of 2:\t\t%ld\n", header.paddedLength);
    printf("Padded observation time:\t%lf\n", header.tsamp * header.paddedLength);
    printf("FFT bin width\t\t\t%lf Hz\n", 1.0 / (header.tsamp * header.paddedLength));

    // compute the time shifts for each channel
    double* timeShifts = (double*) malloc(header.nchans * sizeof(double));
    compute_time_shifts(timeShifts, (double)header.fch1, (double)header.foff, header.nchans, (double)1.0, (double) (1.0 / (header.tsamp * header.paddedLength)));

    int* numCopiesArray = (int*) malloc(header.nchans * sizeof(int));
    // set numCopiesArray to all ones
    for (int i = 0; i < header.nchans; i++) {
        numCopiesArray[i] = 1;
    }

    double* timeShiftDifferences = (double*) malloc((header.nchans-1) * sizeof(double));
    //for (int i = 0; i < header.nchans-1; i++) {
    double gradientRatioSum = 0.0f;
    int totalExtraChannels = 0;
    for (int i = 0; i < header.nchans-1; i++) {
        timeShiftDifferences[i] = timeShifts[i+1] - timeShifts[i];
        gradientRatioSum += (timeShiftDifferences[i] / timeShiftDifferences[0])-1;
        while (gradientRatioSum > 1.0f) {
            gradientRatioSum -= 1.0f;
            totalExtraChannels++;
            numCopiesArray[i]++;
        }
    }

    printf("Total extra channels:\t\t%d\n", totalExtraChannels);

    int numChannelsStretched = header.nchans + totalExtraChannels;
    int numChannelsStretchedPadded = numChannelsStretched;

    // increase numChannelsStretchedPadded to the next power of 2
    int nextPowerOf2 = 1;
    while (nextPowerOf2 < numChannelsStretchedPadded) {
        nextPowerOf2 *= 2;
    }
    numChannelsStretchedPadded = nextPowerOf2;

    printf("Number of channels stretched:\t%d\n", numChannelsStretched);
    printf("Number of channels stretched and padded:\t%d\n", numChannelsStretchedPadded);

    // transpose the data
    uint8_t* transposedData = (uint8_t*) malloc(header.nsamp * header.nchans * sizeof(uint8_t));
    for (int i = 0; i < header.nchans; i++) {
        for (int j = 0; j < header.nsamp; j++) {
            transposedData[i * header.nsamp + j] = hostFilterbank.data[j * header.nchans + i];
        }
    }

    // replace the data with the transposed data
    free(hostFilterbank.data);
    hostFilterbank.data = transposedData;

    // create a new array to hold the stretched and padded data
    uint8_t* stretchedPaddedData = (uint8_t*) malloc(header.nsamp * numChannelsStretchedPadded * sizeof(uint8_t));
    //memset it to zero
    memset(stretchedPaddedData, 0, header.nsamp * numChannelsStretchedPadded * sizeof(uint8_t));

    // copy the data into the stretched and padded array, copying each channel the corresponding number of times as in numCopiesArray
    int stretchedPaddedDataIndex = 0;
    for (int i = 0; i < header.nchans; i++) {
        for (int j = 0; j < numCopiesArray[i]; j++) {
            for (int k = 0; k < header.nsamp; k++) {
                stretchedPaddedData[stretchedPaddedDataIndex] = hostFilterbank.data[i * header.nsamp + k];
                stretchedPaddedDataIndex++;
            }
        }
    }

    // write the stretched and padded data to a csv file, in format: channel, time, intensity
    FILE *csvFile = fopen("stretchedPaddedData.csv", "w");
    for (int i = 0; i < numChannelsStretchedPadded; i++) {
        for (int j = 0; j < header.nsamp; j++) {
            fprintf(csvFile, "%d, %d, %d\n", i, j, stretchedPaddedData[i * header.nsamp + j]);
        }
    }














    free(hostFilterbank.data);
    free(timeShifts);

    // stop timing
    gettimeofday(&end, NULL);
    printf("Total time:\t\t\t%lf s\n", (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0);
    return 0;
}
