
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <chrono>

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA Error in " << __FILE__ << " at line " << __LINE__ << ": " << hipGetErrorString(err); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

#define CHECK_CUBLAS(call) \
    do { \
        hipblasStatus_t stat = call; \
        if (stat != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS Error in " << __FILE__ << " at line " << __LINE__ << ": " << stat; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

    int main() {
        const int N = 4096; // size of the matrix, adjust as needed
        float *A, *B, *C;
        half *A16, *B16, *C16;
    
        hipblasHandle_t handle; // cuBLAS context
        CHECK_CUBLAS(hipblasCreate(&handle)); // Initialize cuBLAS context
    
        CHECK_CUDA(hipMalloc(&A, N * N * sizeof(float)));
        CHECK_CUDA(hipMalloc(&B, N * N * sizeof(float)));
        CHECK_CUDA(hipMalloc(&C, N * N * sizeof(float)));
        CHECK_CUDA(hipMalloc(&A16, N * N * sizeof(half)));
        CHECK_CUDA(hipMalloc(&B16, N * N * sizeof(half)));
        CHECK_CUDA(hipMalloc(&C16, N * N * sizeof(half)));
    
        // Initialize matrices A and B here...
    
        const float alpha = 1.0f; // Moved these before their first usage
        const float beta = 0.0f;
    
        CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
        CHECK_CUBLAS(hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, &alpha, A, N, &beta, B, N, C, N));
        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, A16, HIP_R_16F, N, B16, HIP_R_16F, N, &beta, C16, HIP_R_16F, N, HIP_R_16F, CUBLAS_GEMM_DFALT_TENSOR_OP));
    
        // Record the start time
        auto start = std::chrono::high_resolution_clock::now();
    
        // Do the matrix multiplication
        CHECK_CUBLAS(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, A16, HIP_R_16F, N, B16, HIP_R_16F, N, &beta, C16, HIP_R_16F, N, HIP_R_16F, CUBLAS_GEMM_DFALT_TENSOR_OP));
        CHECK_CUDA(hipDeviceSynchronize());  // Ensure kernel completion before stopping the timer
    
        // Record the end time
        auto end = std::chrono::high_resolution_clock::now();

        std::chrono::duration<double> elapsed = end - start;

        // Calculate throughput in GB/s
        double totalDataGB = 3.0 * N * N * sizeof(half) / (1024.0 * 1024.0 * 1024.0);  // 3 matrices * N * N elements * 2 bytes each
        double throughputGBs = totalDataGB / elapsed.count();

        // Calculate throughput in TFLOPs/s
        double totalFLOPs = (2.0 * N * N * N - N * N) / 1e12;  // 2N^3 - N^2 FLOPs
        double throughputTFLOPs = totalFLOPs / elapsed.count();

        std::cout << "Matrix multiplication took: " << elapsed.count() << " seconds." << std::endl;
        std::cout << "Throughput: " << throughputGBs << " GB/s" << std::endl;
        std::cout << "Performance: " << throughputTFLOPs << " TFLOPs/s" << std::endl;

    
        // Clean up
        CHECK_CUDA(hipFree(A));
        CHECK_CUDA(hipFree(B));
        CHECK_CUDA(hipFree(C));
        CHECK_CUDA(hipFree(A16));
        CHECK_CUDA(hipFree(B16));
        CHECK_CUDA(hipFree(C16));
    
        CHECK_CUBLAS(hipblasDestroy(handle)); // Destroy cuBLAS context
    
        return 0;
    }
